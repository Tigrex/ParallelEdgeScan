
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <time.h>





typedef struct Edge {
	int fromIndex;
	int toIndex;
	int departure;
	int arrival;
} Edge;



/*
********************************************************************************************************
Dynamic integer array
********************************************************************************************************
*/
typedef struct IntArray {
	int* array;
	size_t used;
	size_t size;
} IntArray;

void initIntArray(IntArray* a, size_t initialSize) {
	a->array = (int*)malloc(initialSize* sizeof(int));
	a->used = 0;
	a->size = initialSize;
}

void insertIntArray(IntArray* a, int element) {
	if (a->used == a->size) {
  		a->size *= 2;
		a->array = (int *)realloc(a->array, a->size * sizeof(int));
	}
	a->array[a->used++] = element;
}

void freeIntArray(IntArray* a) {
	free(a->array);
	a->array = NULL;
	a->used = a->size = 0;
}







/*
********************************************************************************************************
Dynamic edge array
********************************************************************************************************
*/
typedef struct EdgeArray {
	Edge* array;
	size_t used;
	size_t size;
} EdgeArray;

void initEdgeArray(EdgeArray* a, size_t initialSize) {
	a->array = (Edge *)malloc(initialSize * sizeof(Edge));
	a->used = 0;
	a->size = initialSize;

	// Initialize all values of the array to 0
	for(int i = 0; i<initialSize; i++) {
		memset(&a->array[i],0,sizeof(Edge));
	}
}

void insertEdgeArray(EdgeArray* a, Edge element) {
	if (a->used == a->size) {
		a->size *= 2;
		a->array = (Edge*)realloc(a->array, a->size * sizeof(Edge));
	}

	// Copy all fields
    a->array[a->used].fromIndex = element.fromIndex;
    a->array[a->used].toIndex = element.toIndex;
    a->array[a->used].departure = element.departure;
    a->array[a->used].arrival = element.arrival;

    a->used++;
}

void freeEdgeArray(EdgeArray* a) {
	free(a->array);
	a->array = NULL;
	a->used = a->size = 0;
}








/*
********************************************************************************************************
Function Definitions
********************************************************************************************************
*/
int stringToInt(char* str, int len);
int compare(const void* a, const void* b);

// berlin
int numOfBatches = 1495;
int numOfEdges = 1209980;
int numOfVertices = 12746;
char file1[] = "e:\\data\\berlin.txt.meta";
char file2[] = "e:\\data\\berlin.txt.edges";

// wikipedia-growth
// int numOfBatches = 700;
// int numOfEdges = 39953145;
// int numOfVertices = 1870709;
// char file1[] = "e:\\data\\out.wikipedia-growth.meta";
// char file2[] = "e:\\data\\out.wikipedia-growth.edges";

// munmun_digg_reply
// int numOfBatches = 30;
// int numOfEdges = 86203;
// int numOfVertices = 30360;
// char file1[] = "e:\\data\\out.munmun_digg_reply.meta";
// char file2[] = "e:\\data\\out.munmun_digg_reply.edges";

// loans
// int numOfBatches = 59;
// int numOfEdges = 3343284;
// int numOfVertices = 89269;
// char file1[] = "e:\\data\\out.prosper-loans.meta";
// char file2[] = "e:\\data\\out.prosper-loans.edges";

// digg-friends
// int numOfBatches = 4623;
// int numOfEdges = 1731653;
// int numOfVertices = 279630;
// char file1[] = "e:\\data\\out.digg-friends.meta";
// char file2[] = "e:\\data\\out.digg-friends.edges";


int numOfRuns = 100;

bool printResult = false;

/*
********************************************************************************************************
Entry point
********************************************************************************************************
*/
int main() {

	/*
	********************************************************************************************************
	Read meta file
	********************************************************************************************************
	*/

	printf("\nReading file %s...\n", file1);


	FILE *fp;
	fp = fopen(file1,"r"); // read mode
 
	if(fp == NULL) {
		perror("Error while opening the file.\n");
		exit(EXIT_FAILURE);
	}


	int* batchSizes = (int*)malloc(numOfBatches* sizeof(int));


	char line[256];
	int count = 0;
	while (fgets(line, sizeof(line), fp) != NULL) {
        /* note that fgets don't strip the terminating \n, checking its
           presence would allow to handle lines longer that sizeof(line) */

		int size = stringToInt(line, strlen(line) - 1);
		batchSizes[count] = size;
		count++;

		// printf("New batch size added: %d...\n", size);
	}
    
	
	fclose(fp);




	/*
	********************************************************************************************************
	Read data file
	********************************************************************************************************
	*/

	printf("\nReading file %s...\n", file2);

	fp = fopen(file2,"r"); // read mode
 
	if(fp == NULL) {
		perror("Error while opening the file.\n");
		exit(EXIT_FAILURE);
	}
 

	Edge* edges = (Edge *)malloc(numOfEdges * sizeof(Edge));
	// Initialize all values of the array to 0
	// for(int i = 0; i < numOfEdges; i++) {
	// 	memset(&(edges[i]),0,sizeof(Edge));
	// }

	count = 0;
	while (fgets(line, sizeof(line), fp) != NULL) {
        /* note that fgets don't strip the terminating \n, checking its
           presence would allow to handle lines longer that sizeof(line) */
		
		char copy[256] = "";
		strncpy(copy, line, strlen(line));

		char* parts;
		parts = strtok(copy," ");

		if (parts != NULL) {
			edges[count].fromIndex = stringToInt(parts, strlen(parts));
			parts = strtok(NULL, " ");
			edges[count].toIndex = stringToInt(parts, strlen(parts));
			parts = strtok(NULL, " ");
			edges[count].departure = stringToInt(parts, strlen(parts));
			parts = strtok(NULL, " ");
			edges[count].arrival = stringToInt(parts, strlen(parts) - 1);
		}

		count++;

	}

	fclose(fp);



	

	/*
	********************************************************************************************************
	Get shortest paths
	********************************************************************************************************
	*/

	printf("\nCalculating shortest paths...\n");

	int* labels = (int*)malloc(numOfVertices * sizeof(int));
	int departureTime = 0;

	clock_t begin, end;
	double time_spent;
	begin = clock();

	for (int r = 0; r < numOfRuns; r++) {

		// Initialize labels
		for (int i = 0; i < numOfVertices; i++) {
			labels[i] = INT_MAX;
		}

		labels[r] = departureTime;

		int start = 0;
		for (int i = 0; i < numOfBatches; i++) {
			int size = batchSizes[i];

			for (int j = start; j < start + size; j++) {
				Edge e = edges[j];
				int fromIndex = e.fromIndex;
				int toIndex = e.toIndex;
				int departure = e.departure;
				int arrival = e.arrival;

				if (departure >= labels[fromIndex]) {

					if (arrival < labels[toIndex]) {

						labels[toIndex] = arrival;

					}

				}

			}

			start = start + size;

		}

		if (printResult) {
			int numOfReachable = 0;
			for (int i = 0; i < numOfVertices; i++) {
				if (labels[i] != INT_MAX) {
					numOfReachable++;
				}
			}
			printf("Number of reachable vertices: %d...\n", numOfReachable);

		}

	}

	end = clock();
	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("Program running time: %f\n", time_spent);

	

	/*
	********************************************************************************************************
	Cleanup
	********************************************************************************************************
	*/


	free(labels);
	free(batchSizes);
	free(edges);

	return 0;
}



int stringToInt(char* str, int len) {

	int size = 0;

	for(int i = 0; i < len; i++) {
		size = size * 10 + (str[i] - '0');

	}
	return size;
}
		

int compare(const void* a, const void* b) {
     int int_a = * ( (int*) a );
     int int_b = * ( (int*) b );

     if ( int_a == int_b ) return 0;
     else if ( int_a < int_b ) return -1;
     else return 1;
}







 

