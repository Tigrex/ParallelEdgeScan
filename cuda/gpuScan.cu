
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <time.h>



#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



typedef struct Edge {
	int fromIndex;
	int toIndex;
	int departure;
	int arrival;
} Edge;

/*
********************************************************************************************************
Cuda functions
********************************************************************************************************
*/
__global__ 
void edgeScan(int n, unsigned int* labels, Edge* edges, int offset) {

	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < n) {

		Edge e = edges[offset + i];

		int fromIndex = e.fromIndex;
		int toIndex = e.toIndex;
		int departure = e.departure;
		int arrival = e.arrival;

		if (departure >= labels[fromIndex]) {

			atomicMin(&(labels[toIndex]), arrival);

		}

	}

}



__global__
void initLabels(int n, unsigned int* labels) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < n) {
		labels[i] = INT_MAX;
	}
}

__global__
void setSource(unsigned int* labels, int source, int time) {
	labels[source] = time;
}



/*
********************************************************************************************************
Function Definitions
********************************************************************************************************
*/
int stringToInt(char* str, int len);
int compare(const void* a, const void* b);


// berlin
int numOfBatches = 1495;
int numOfEdges = 1209980;
int numOfVertices = 12746;
char file1[] = "e:\\data\\berlin.txt.meta";
char file2[] = "e:\\data\\berlin.txt.edges";

// wikipedia-growth
// int numOfBatches = 700;
// int numOfEdges = 39953145;
// int numOfVertices = 1870709;
// char file1[] = "e:\\data\\out.wikipedia-growth.meta";
// char file2[] = "e:\\data\\out.wikipedia-growth.edges";

// munmun_digg_reply
// int numOfBatches = 30;
// int numOfEdges = 86203;
// int numOfVertices = 30360;
// char file1[] = "e:\\data\\out.munmun_digg_reply.meta";
// char file2[] = "e:\\data\\out.munmun_digg_reply.edges";

// loans
// int numOfBatches = 59;
// int numOfEdges = 3343284;
// int numOfVertices = 89269;
// char file1[] = "e:\\data\\out.prosper-loans.meta";
// char file2[] = "e:\\data\\out.prosper-loans.edges";

// digg-friends
// int numOfBatches = 4623;
// int numOfEdges = 1731653;
// int numOfVertices = 279630;
// char file1[] = "e:\\data\\out.digg-friends.meta";
// char file2[] = "e:\\data\\out.digg-friends.edges";


int numOfRuns = 100;

const int max_thread = 512;

bool printResult = false;

/*
********************************************************************************************************
Entry point
********************************************************************************************************
*/
int main() {

	/*
	********************************************************************************************************
	Read meta file
	********************************************************************************************************
	*/

	printf("\nReading file %s...\n", file1);


	FILE *fp;
	fp = fopen(file1,"r"); // read mode
 
	if(fp == NULL) {
		perror("Error while opening the file.\n");
		exit(EXIT_FAILURE);
	}


	int* batchSizes = (int*)malloc(numOfBatches* sizeof(int));


	char line[256];
	int count = 0;
	while (fgets(line, sizeof(line), fp) != NULL) {
        /* note that fgets don't strip the terminating \n, checking its
           presence would allow to handle lines longer that sizeof(line) */

		int size = stringToInt(line, strlen(line) - 1);
		batchSizes[count] = size;
		count++;

		// printf("New batch size added: %d...\n", size);
	}
    
	
	fclose(fp);



	/*
	********************************************************************************************************
	Read data file
	********************************************************************************************************
	*/

	printf("\nReading file %s...\n", file2);

	fp = fopen(file2,"r"); // read mode
 
	if(fp == NULL) {
		perror("Error while opening the file.\n");
		exit(EXIT_FAILURE);
	}
 

	Edge* edges = (Edge *)malloc(numOfEdges * sizeof(Edge));
	// Initialize all values of the array to 0
	// for(int i = 0; i < numOfEdges; i++) {
	// 	memset(&(edges[i]),0,sizeof(Edge));
	// }

	count = 0;
	while (fgets(line, sizeof(line), fp) != NULL) {
        /* note that fgets don't strip the terminating \n, checking its
           presence would allow to handle lines longer that sizeof(line) */
		
		char copy[256] = "";
		strncpy(copy, line, strlen(line));

		char* parts;
		parts = strtok(copy," ");

		if (parts != NULL) {
			edges[count].fromIndex = stringToInt(parts, strlen(parts));
			parts = strtok(NULL, " ");
			edges[count].toIndex = stringToInt(parts, strlen(parts));
			parts = strtok(NULL, " ");
			edges[count].departure = stringToInt(parts, strlen(parts));
			parts = strtok(NULL, " ");
			edges[count].arrival = stringToInt(parts, strlen(parts) - 1);
		}

		count++;

	}

	fclose(fp);


	/*
	********************************************************************************************************
	Get shortest paths
	********************************************************************************************************
	*/

	// GPU implementation
	unsigned int* labels = (unsigned int*)malloc(numOfVertices * sizeof(unsigned int));

	unsigned int* gpuLabels;
	Edge* gpuEdges;
	

	const int labelSize = numOfVertices * sizeof(unsigned int);
	const int edgeSize = numOfEdges * sizeof(Edge);
	

	gpuErrchk(hipMalloc((void**)&gpuLabels, labelSize));
	gpuErrchk(hipMalloc((void**)&gpuEdges, edgeSize));
	

	gpuErrchk(hipMemcpy(gpuEdges, edges, edgeSize, hipMemcpyHostToDevice));
	gpuErrchk(hipDeviceSynchronize());

	clock_t begin, end;
	double time_spent;
	begin = clock();

	for (int i = 0; i < numOfRuns; i++) {
		// Initialize labels

		initLabels<<<numOfVertices / max_thread + 1, max_thread>>>(numOfVertices, gpuLabels);
		gpuErrchk(hipPeekAtLastError());

		gpuErrchk(hipDeviceSynchronize());

		// end = clock();
		// time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		// printf("Up to init label running time: %f\n", time_spent);


		setSource<<<1, 1>>>(gpuLabels, i, 0);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		// end = clock();
		// time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		// printf("Up to set source running time: %f\n", time_spent);


		// Start edge scan
		int offset = 0;
		for (int j = 0; j < numOfBatches; j++) {
			int size = batchSizes[j];

			edgeScan<<<size / max_thread + 1, max_thread>>>(size, gpuLabels, gpuEdges, offset);
			gpuErrchk(hipPeekAtLastError());
			
			gpuErrchk(hipDeviceSynchronize());

			offset = offset + size;
		}

		// end = clock();
		// time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		// printf("Up to GPU computing running time: %f\n", time_spent);



		gpuErrchk(hipMemcpy(labels, gpuLabels, labelSize, hipMemcpyDeviceToHost));
		gpuErrchk(hipDeviceSynchronize());

		if (printResult) {
			int numOfReachable = 0;
			for (int j = 0; j < numOfVertices; j++) {
				//printf("Label[%d]=%d\n", j, labels[j]);
				if (labels[j] != INT_MAX) {
					numOfReachable++;
				}
			}
			printf("Number of reachable vertices: %d...\n", numOfReachable);

		}

	}

	end = clock();
	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("Total running time: %f\n", time_spent);

	


	// Clean up GPU memory
	hipFree(gpuLabels);
	hipFree(gpuEdges);
	//cudaFree(gpuBatchSizes);


	hipDeviceSynchronize();



	/*
	********************************************************************************************************
	Cleanup
	********************************************************************************************************
	*/


	free(labels);

	free(batchSizes);
	free(edges);

	return 0;
}



int stringToInt(char* str, int len) {

	int size = 0;

	for(int i = 0; i < len; i++) {
		size = size * 10 + (str[i] - '0');

	}
	return size;
}
		

int compare(const void* a, const void* b) {
     int int_a = * ( (int*) a );
     int int_b = * ( (int*) b );

     if ( int_a == int_b ) return 0;
     else if ( int_a < int_b ) return -1;
     else return 1;
}







 

